#include "hip/hip_runtime.h"
#include <wb.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define NUM_BINS 4096
#define BLOCK_SIZE 512 

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
	bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
			file, line);
		if (abort)
			exit(code);
	}
}

__global__ void histogram(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {
	//@@ Write the kernel that computes the histogram
	//@@ Make sure to use the privitization technique
	
	__shared__ unsigned int private_histo[NUM_BINS];

	for (unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx += BLOCK_SIZE) { //reference to line 3 from figure 9.10 from pg 211 of the textbook.
		private_histo[binIdx] = 0;
	}

	__syncthreads();

	int tid = threadIdx.x + blockIdx.x * blockDim.x;	//reference PPT12 Slides 45-47
	int stride = blockDim.x * gridDim.x;

	while(tid < num_elements) {
		int numberValue = input[tid];
		if (numberValue >= 0 && numberValue < num_bins) {
			atomicAdd(&(private_histo[numberValue]), 1);
		}
		tid += stride;
	}
	
	__syncthreads();

	for (unsigned int binIdx = threadIdx.x; binIdx < NUM_BINS; binIdx += BLOCK_SIZE) { //reference to line 9 from figure 9.10 from pg 211 of the textbook.
		atomicAdd(&(bins[binIdx]), private_histo[binIdx]);
	}
}

__global__ void saturate(unsigned int *bins, unsigned int num_bins) {
	//@@ Write the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)

	for (int i = 0; i < num_bins; i++) {  //simple function for 127 value cap.
		if (bins[i] > 127) {
			bins[i] = 127;
		}
	}

}

int main(int argc, char *argv[]) {
	wbArg_t args;
	int inputLength;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;
	int numBlocks;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0), &inputLength, "Integer");
	hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
	numBlocks = (float(inputLength - 1)) / BLOCK_SIZE + 1;
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating device memory");
	//@@ Allocate device memory here
	CUDA_CHECK(hipDeviceSynchronize());

	hipMalloc((void**)&deviceInput, inputLength * sizeof(float));
	hipMalloc((void**)&deviceBins, NUM_BINS * sizeof(float));
		wbTime_stop(GPU, "Allocating device memory");

	wbTime_start(GPU, "Copying input host memory to device");
	//@@ Copy input host memory to device
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(float), hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input host memory to device");

	wbTime_start(GPU, "Clearing the bins on device");
	//@@ zero out the deviceBins using hipMemset() 
	hipMemset(deviceBins, 0, NUM_BINS * sizeof(float));
	wbTime_stop(GPU, "Clearing the bins on device");

	//@@ Initialize the grid and block dimensions here
	dim3 GridDim(numBlocks, 1, 1);
	dim3 BlockDim(BLOCK_SIZE, 1, 1);

	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Invoke kernels: first call histogram kernel and then call saturate kernel

	histogram <<< GridDim, BlockDim >>> (deviceInput, deviceBins, inputLength, NUM_BINS);

	CUDA_CHECK(hipDeviceSynchronize());

	saturate <<< GridDim, BlockDim >>> (deviceBins, NUM_BINS);

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output device memory to host");
	
	//@@ Copy output device memory to host


	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(float), hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying output device memory to host");

	wbTime_start(GPU, "Freeing device memory");
	//@@ Free the device memory here
	hipFree(deviceInput);
	hipFree(deviceBins);
	wbTime_stop(GPU, "Freeing device memory");

	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);
	return 0;
}
